
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 100
#define BLOCKSIZE 256

double get_clock() {
 struct timeval tv; int ok;
 ok = gettimeofday(&tv, (void *) 0);
 if (ok<0) { printf("gettimeofday error"); }
 return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void recursive_doubling(int *input, int *output, int stride) {
   int i = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
   __syncthreads();
    if (i < SIZE) {
       if(i<stride){
		output[i]=input[i];
	}else{
		output[i] = input[i]+input[i-stride];
	}
    }
}
int main(void) {
    // Allocate host memory
    int* h_input = (int*)malloc(sizeof(int) * SIZE);
    int* h_output = (int*)malloc(sizeof(int) * SIZE);

    // Initialize input on host
    for (int i = 0; i < SIZE; i++) {
        h_input[i] = 1;
    }

    // Allocate device memory
    int* d_input;
    int* d_output;
    hipMalloc((void**)&d_input, sizeof(int) * SIZE);
    hipMalloc((void**)&d_output, sizeof(int) * SIZE);

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    
    double t0,t1;
    t0=get_clock();
     for (int stride = 1; stride < SIZE; stride *= 2) {
     	int numSize = (SIZE + BLOCKSIZE -1)/BLOCKSIZE;
        recursive_doubling<<<numSize, BLOCKSIZE>>>(d_input, d_output, stride);

        int *temp = d_output;
        d_output = d_input;
        d_input = temp;
    }
    hipDeviceSynchronize();
    t1=get_clock();
    
    // Copy output data back to host
    hipMemcpy(h_output, d_input, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < SIZE; i++) {
      printf("%d ", h_output[i]);
    }
    printf("\n");

    printf("time per call: %f ns\n", (1000000000.0*(t1-t0)) );

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}